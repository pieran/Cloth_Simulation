/** File:		sph_kernel.cu
 ** Author:		Dongli Zhang
 ** Contact:	dongli.zhang0129@gmail.com
 **
 ** Copyright (C) Dongli Zhang 2013
 **
 ** This program is free software;  you can redistribute it and/or modify
 ** it under the terms of the GNU General Public License as published by
 ** the Free Software Foundation; either version 2 of the License, or
 ** (at your option) any later version.
 **
 ** This program is distributed in the hope that it will be useful,
 ** but WITHOUT ANY WARRANTY;  without even the implied warranty of
 ** MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See
 ** the GNU General Public License for more details.
 **
 ** You should have received a copy of the GNU General Public License
 ** along with this program;  if not, write to the Free Software 
 ** Foundation, 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA
 */

#ifndef __SPHKERNEL_CU__
#define __SPHKERNEL_CU__

#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/device_ptr.h>
#include <thrust/for_each.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/sort.h>

#include <vector>
#include <list>

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#include "sph3d_system.h"

__constant__ SysParam dParam;

#endif
